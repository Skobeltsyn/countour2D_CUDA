#include "hip/hip_runtime.h"
#include "toolpath.h"

void handle_error_cuda(char *line){
    hipError_t err = hipGetLastError();
    if( hipSuccess != err){
        printf("%s) Cuda error: %s.\n", line, hipGetErrorString( err) );
        exit(0);
    }                        
}

__device__ double dMin (double a, double b) {
	double output = a;
	if (a>b) output = b;
	if (b>a) output = a;
	return output;
}

__device__ double dMax (double a, double b) {
	double output = a;
	if (a>b) output = a;
	if (b>a) output = b;
	return output;
}
__global__ void checkIntersection(double *lineX0, double *lineY0, double *lineX1, double *lineY1, short *isThereIntersection, long int numOfLines, long int lineId, double *catchedIntersectionX, double *catchedIntersectionY, double eps){
	int idx = blockDim.x* blockIdx.x + threadIdx.x;
	if (idx < numOfLines) {
		if (idx != lineId) {
			isThereIntersection[idx] = 0;
			if ((lineX0[idx] != lineX1[idx]) && (lineX0[lineId] != lineX1[lineId])) {
				double line1_k = (lineY1[idx] - lineY0[idx]) 		/ (lineX1[idx] - lineX0[idx]);
				double line2_k = (lineY1[lineId] - lineY0[lineId]) 	/ (lineX1[lineId] - lineX0[lineId]);

				double line1_y0 = lineY0[idx] 		- line1_k * lineX0[idx];
				double line2_y0 = lineY0[lineId] 	- line2_k * lineX0[lineId];
				if (line2_k != line1_k) {
					double intersection_x = (line1_y0 - line2_y0) / (line2_k - line1_k);
					double intersection_y = line1_y0 + line1_k * intersection_x;
					if ((intersection_x >= dMin(lineX0[idx], 	lineX1[idx]) - eps) 			&&
						(intersection_x <= dMax(lineX0[idx], 	lineX1[idx]) + eps) 			&&
						(intersection_x >= dMin(lineX0[lineId],  lineX1[lineId]) - eps) 		&&	
						(intersection_x <= dMax(lineX0[lineId], 	lineX1[lineId]) + eps)	&&
						(intersection_y >= dMin(lineY0[idx], lineY1[idx]) - eps)				&&
						(intersection_y <= dMax(lineY0[idx], lineY1[idx]) + eps)				&&
						(intersection_y >= dMin(lineY0[lineId], lineY1[lineId]) - eps)		&&
						(intersection_y <= dMax(lineY0[lineId], lineY1[lineId]) + eps)		
						)
					   
					{
						isThereIntersection[idx] = 1;
						catchedIntersectionX[idx] = intersection_x;
						catchedIntersectionY[idx] = intersection_y;
					}
				}
			}
		}
	}
}
#include "detectOuter.h"

short  sign_host(double x) {
	short output = 0;
	if (x<0) output = -1;
	if (x>0) output = 1;
	return output;
}

double vetctorMult_host(double ax, double ay, double bx, double by) {
	return ax * by - ay * bx;
}

int checkPoint_host(double dotX, double dotY, double *polygonX, double *polygonY, long int *profileLength, long int *profileStartIndex, long int numOfProfiles, long int profileId){
	long int startIndex = profileStartIndex[profileId];
	double abX = polygonX[startIndex+1] - polygonX[startIndex];
	double abY = polygonY[startIndex+1] - polygonY[startIndex];
	double apX = dotX - polygonX[startIndex];
	double apY = dotY - polygonY[startIndex];
	double bpX = dotX - polygonX[startIndex+1];
	double bpY = dotY - polygonY[startIndex+1];
	short pointIn = 1;
	short currSign = sign_host(vetctorMult_host(abX,abY,apX,apY));
	for (long int i = startIndex + 1; i < startIndex + profileLength[profileId] ; i++){
		short prevSign = currSign;
		if (prevSign == 0) {
			if 	(((bpX*bpX + bpY*bpY) > (abX*abX + abY * abY)) ||
				((apX*apX + apY*apY) > (abX*abX + abY * abY))) 
			{
				 pointIn = 0;
				 break;
			}
		}
		
		if (i != startIndex + profileLength[profileId] - 1 ){
			abX = polygonX[i+1] - polygonX[i];
			abY = polygonY[i+1] - polygonY[i];
			apX = dotX - polygonX[i];
			apY = dotY - polygonY[i];
			bpX = dotX - polygonX[i+1];
			bpY = dotY - polygonY[i+1];
			currSign = sign_host(vetctorMult_host(abX,abY,apX,apY));
		} else {
			abX = polygonX[startIndex] - polygonX[i];
			abY = polygonY[startIndex] - polygonY[i];
			apX = dotX - polygonX[i];
			apY = dotY - polygonY[i];
			bpX = dotX - polygonX[startIndex];
			bpY = dotY - polygonY[startIndex];
			currSign = sign_host(vetctorMult_host(abX,abY,apX,apY));
		}
		
		if (currSign * prevSign < 0) {pointIn = 0; break;}
	}
	return pointIn;
}

int main(int argc, char const *argv[]){
	#include "inputData"
	#include "detectLines"
	#include "detectProfiles"
	#include "copyDataToDevice"
	#include "printLines"
	#include "detectIntersections"
	// #include "printIntersections"
	#include "copyIntersectionsToCUDA"
	// for (int i = 0; i < numOfPoints; i++){
	// 	printf("%lf %lf %ld\n",polygonX_host[i],polygonY_host[i],polygonId_host[i] );
	// }
	// double *finalIntersectionsX = (double *)malloc(sizeof(double));
	// double *finalIntersectionsY = (double *)malloc(sizeof(double));
	// long int tempNumOfIntersections = 0;
	// for (long int i = 0; i < numOfIntersections; i++) {
	// 	int numOfRepeates = 0;
	// 	for (int j = i+1; j < numOfIntersections; j++){
	// 		if ((fabs(intersectionsX_host[i] - intersectionsX_host[j]) <= eps) &&
	// 			(fabs(intersectionsY_host[i] - intersectionsY_host[j]) <= eps)
	// 			)
	// 		numOfRepeates++;
	// 		intersectionsY_host[j] = 
	// 	}
	// 	if ()
	// }
	#include "detectOuter"
	
	
	
	FILE *outer = fopen("intersections.txt","w");
	for (long int i = 0; i < numOfIntersections; i++) {
		// pointIsOuter_host[i]=1;
		if (pointIsOuter_host[i] >= 1) {
			numOfEdgePoints++;
			fprintf(outer, "%lf,%lf\n",intersectionsX_host[i],intersectionsY_host[i] );
		}
	}
	// float radius = 5;
	// for (float angle = 0; angle < 360; angle+=0.001){
	// 	float tempX = radius * cos(angle * 3.14 / 180);
	// 	float tempY = radius * sin(angle * 3.14 / 180);
	// 	tempX = (float)(random() % 2000)/100 - 8;
	// 	tempY = (float)(random() % 5000)/100 - 20;
	// 	for (int i = 0; i < numOfProfiles; i++) {
	// 		if (checkPoint_host(tempX, tempY, polygonX_host, polygonY_host, profileLength_host, profileStartIndex_host, numOfProfiles, i) == 1) {
	// 			int numOfIntersectedProfiles = 0;
	// 			float rad = 0.1;
	// 			for (float x = 0; x < 360; x+=20) {
	// 				double tx = tempX + rad * cos(x*3.14/180);
	// 				double ty = tempY + rad * sin(x*3.14/180);
	// 				if (checkPoint_host(tx, ty, polygonX_host, polygonY_host, profileLength_host, profileStartIndex_host, numOfProfiles, i) == 1) {
	// 					numOfIntersectedProfiles++;
	// 				}
	// 			}
	// 			if (numOfIntersectedProfiles<=1) fprintf(outer, "%lf,%lf\n",tempX,tempY );
	// 		}
	// 	}
	// }
	
	fclose(outer);

	hipFree(polygonX_device);
	hipFree(polygonY_device);
	hipFree(polygonId_device);
	hipFree(lineX0_device);
	hipFree(lineY0_device);
	hipFree(lineX1_device);
	hipFree(lineY1_device);
	hipFree(catchedIntersectionX_device);
	hipFree(catchedIntersectionY_device);
	hipFree(isThereIntersection_device);
	hipFree(intersectionsX_device);
	hipFree(intersectionsY_device);
	hipFree(profileStartIndex_device);
	hipFree(profileLength_device);
	hipFree(pointIsOuter_device);

	free(pointIsOuter_host);
	free(profileLength_host);
	free(profileStartIndex_host);
	free(isThereIntersection_host);
	free(lineX0_host);
	free(lineY0_host);
	free(lineX1_host);
	free(lineY1_host);
	free(catchedIntersectionX_host);
	free(catchedIntersectionY_host);
	free(polygonX_host );
	free(polygonY_host );
	free(polygonId_host);
	free(intersectionsX_host);
	free(intersectionsY_host);


	printf("numOfPoints=%ld\n",numOfPoints );
	printf("numOfLines=%ld\n",numOfLines );
	printf("numOfProfiles=%ld\n",numOfProfiles );
	printf("numOfIntersections=%ld\n",numOfIntersections );
	printf("numOfEdgePoints=%ld\n", numOfEdgePoints );
	return 0;
}